
#include <hip/hip_runtime.h>
#include <iostream>

inline void CHECK_CUDA_ERROR(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error : " << hipGetErrorString(err) << " at " << __FILE__ << " : " << __LINE__ << std::endl;
        exit(EXIT_FAILURE);
    }
}