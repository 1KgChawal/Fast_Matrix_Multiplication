
#include <hip/hip_runtime.h>
template <typename T>
__global__ void dkernal(T* d_M, T* d_M1, T* d_M2, int rows, int cols, int r_c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        d_M[i * cols + j] = 0;
        for (int k = 0; k < r_c; k++) {
            d_M[i * cols + j] += d_M1[i * r_c + k] * d_M2[k * cols + j];
        }
    }
}