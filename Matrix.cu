#include "hip/hip_runtime.h"
#include "Matrix.h"

template <typename T>
Matrix<T>::Matrix() {
    rows = 0;
    cols = 0;
    vector = nullptr;
}
template <typename T>
Matrix<T>::Matrix(int r, int c) : rows(r), cols(c) {
    vector = new T[r * c];
}
template <typename T>
T& Matrix<T>::operator()(int r, int c) {
    return vector[r * cols + c];
}
template <typename T>
const T& Matrix<T>::operator()(int r, int c) const {
    return vector[r * cols + c];
}
template <typename T>
T* Matrix<T>::data() {
    return vector;
}
template <typename T>
int Matrix<T>::getRows() {
    return rows;
}
template <typename T>
int Matrix<T>::getCols() {
    return cols;
}
template <typename T>
unsigned int Matrix<T>::bytes() {
    return rows * cols * sizeof(T);
}
template<typename T>
void Matrix<T>::print(){
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            std::cout<<(*this)(i,j)<<" ";
        }
        std::cout<<"\n";
    }
}
template <typename T>
Matrix<T> Matrix<T>::operator*(Matrix<T>& M) {
    return multiply(*this, M);
}
template <typename T>
__global__ void dkernal(T* d_M, T* d_M1, T* d_M2, int rows, int cols, int r_c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        d_M[i * cols + j] = 0;
        for (int k = 0; k < r_c; k++) {
            d_M[i * cols + j] += d_M1[i * r_c + k] * d_M2[k * cols + j];
        }
    }
}
template <typename T>
Matrix<T> Matrix<T>::multiply(Matrix<T>& M1, Matrix<T>& M2) {
    Matrix M(M1.getRows(), M2.getCols());
    int k = M1.getCols();
    T *d_M1, *d_M2, *d_M;
    hipMalloc(&d_M1, M1.bytes());
    hipMalloc(&d_M2, M2.bytes());
    hipMalloc(&d_M, M.bytes());
    hipMemcpy(d_M1, M1.data(), M1.bytes(), hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2.data(), M2.bytes(), hipMemcpyHostToDevice);
    dim3 block(16, 16);
    dim3 grid((M.getRows() + blockDim.x - 1) / blockDim.x, (M.getCols() + blockDim.y - 1) / blockDim.y);
    dkernal<<<grid, block>>>(d_M, d_M1, d_M2, M.getRows(), M.getCols(), k);
    hipMemcpy(M.data(), d_M, M.bytes(), hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_M1);
    hipFree(d_M2);
    return M;
}