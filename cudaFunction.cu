#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaFunction.h.cu"
#include "Error.cu"

template <typename T>
__global__ void dkernal(T* d_M, T* d_M1, T* d_M2, int rows, int cols, int r_c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        d_M[i * cols + j] = 0;
        for (int k = 0; k < r_c; k++) {
            d_M[i * cols + j] += d_M1[i * r_c + k] * d_M2[k * cols + j];
        }
    }
}

template<typename T>
void cudaMultiply(T* M1,T* M2,T* M,int rows, int cols,int r_c){
    unsigned int size1=rows*r_c*sizeof(T);
    unsigned int size2=r_c*cols*sizeof(T);
    unsigned int size=rows*cols*sizeof(T);
    T *d_M1,*d_M2,*d_M;
    CHECK_CUDA_ERROR(hipMalloc(&d_M,size));
    CHECK_CUDA_ERROR(hipMalloc(&d_M1,size1));
    CHECK_CUDA_ERROR(hipMalloc(&d_M2,size2));
    CHECK_CUDA_ERROR(hipMemcpy(d_M1,M1,size1,hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_M2,M2,size2,hipMemcpyHostToDevice));
    dim3 block(16,16);
    dim3 grid((rows+block.x-1)/block.x,(cols+block.y-1)/block.y);
    dkernal<<<grid,block>>>(d_M,d_M1,d_M2,rows,cols,r_c);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipMemcpy(M,d_M,size,hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipFree(d_M));
    CHECK_CUDA_ERROR(hipFree(d_M1));
    CHECK_CUDA_ERROR(hipFree(d_M2));
}
