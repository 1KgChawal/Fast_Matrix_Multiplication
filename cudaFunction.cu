#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaFunction.h.cu"

template <typename T>
__global__ void dkernal(T* d_M, T* d_M1, T* d_M2, int rows, int cols, int r_c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        d_M[i * cols + j] = 0;
        for (int k = 0; k < r_c; k++) {
            d_M[i * cols + j] += d_M1[i * r_c + k] * d_M2[k * cols + j];
        }
    }
}

template<typename T>
void cudaMultiply(T* M1,T* M2,T* M,int rows, int cols,int r_c){
    unsigned int size1=rows*r_c*sizeof(T);
    unsigned int size2=r_c*cols*sizeof(T);
    unsigned int size=rows*cols*sizeof(T);
    T *d_M1,*d_M2,*d_M;
    hipMalloc(&d_M,size);
    hipMalloc(&d_M1,size1);
    hipMalloc(&d_M2,size2);
    hipMemcpy(d_M1,M1,size1,hipMemcpyHostToDevice);
    hipMemcpy(d_M2,M2,size2,hipMemcpyHostToDevice);
    dim3 block(16,16);
    dim3 grid((rows+block.x-1)/block.x,(cols+block.y-1)/block.y);
    dkernal<<<grid,block>>>(d_M,d_M1,d_M2,rows,cols,r_c);
    hipMemcpy(M,d_M,size,hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_M1);
    hipFree(d_M2);
}
