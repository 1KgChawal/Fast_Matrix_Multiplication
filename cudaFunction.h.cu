
#include <hip/hip_runtime.h>
#include <iostream>

template <typename T>
__global__ void dkernal(T* d_M, T* d_M1, T* d_M2, int rows, int cols, int r_c);

template<typename T>
void cudaMultiply(T* M1,T* M2,T* M,int rows, int cols,int r_c);