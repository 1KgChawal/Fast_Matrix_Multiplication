#include "hip/hip_runtime.h"
#include <iostream>

template <typename T>
class Matrix {
   private:
    int rows, cols;
    T* vector;

   public:
    Matrix() {
        rows = 0;
        cols = 0;
        vector = nullptr;
    }
    Matrix(int r, int c) : rows(r), cols(c) {
        vector = new T[r * c];
    }
    T& operator()(int r, int c) {
        return vector[r * c + c];
    }
    const T& operator(int r, int c) const {
        return vector[r * cols + c];
    }
    T* data {
        return vector;
    }
    int getRows() {
        return rows;
    }
    int getCols() {
        return cols;
    }
    unsigned int bytes() {
        return r * c * sizeof(T);
    }
    Matrix<T> operator*(Matrix<T>& M) {
        return multiply(*this, M);
    }
    Matrix<T> multiply(Matrix<T>& M1, Matrix<T>& M2) {
        Matrix M(M1.getRows(), M2.getCols());
        int k = M1.getCols();
        T *d_M1, d_M2, d_M;
        hipMalloc(&d_M1, M1.bytes());
        hipMalloc(&d_M2, M2.bytes());
        hipMalloc(&d_M, M.bytes());
        hipMemcpy(d_M1, M1.data(), M1.bytes(), hipMemcpyHostToDevice);
        hipMemcpy(d_M2, M2.data(), M2.bytes(), hipMemcpyHostToDevice);
        dim3 block(16, 16);
        dim3 grid((M.getRows() + blockDim.x - 1) / blockDim.x, (M.getCols() + blockDim.y - 1) / blockDim.y);
        dkernal<<<grid,block>>>(d_M,d_M1,d_M2,M.getRows(),M.getCols(),k);
        hipMemcpy(M.data(),d_M,M.bytes(),hipMemcpyDeviceToHost);
        return M;
    }
};

template <typename T>
__global__ void dkernal(T* d_M, T* d_M1, T* d_M2, int rows, int cols, int r_c) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if(i<rows&&j<cols){
        d_M[i*cols+j]=0;
        for(int k=0;k<r_c;k++){
            d_M[i*cols+j]+=d_M1[i*r_c+k]+d_M2[k*cols+j];
        }
    }
}