#include <iostream>

template<typename T>
class Matrix{
    private:
    int rows,cols;
    T* vector;
    public:
    Matrix(){
        rows=0;
        cols=0;
        vector=nullptr;
    }
    Matrix(int r,int c) : rows(r), cols(c){
        vector=new T [r*c];
    }
    T& operator()(int r,int c){
        return vector[r*c+c];
    }
    const T& operator(int r,int c) const{
        return vector[r*cols+c];
    }
    T* data{
        return vector;
    }
    int getRows(){
        return rows;
    }
    int getCols(){
        return cols;
    }
    unsigned int bytes(){
        return r*c*sizeof(T);
    }
    Matrix<T> operator*(Matrix<T>& M){
        return multiply(*this,M);
    } 
    Matrix<T> multiply(Matrix<T>& M1,Matrix<T>& M2){
        Matrix M(M1.getRows(),M2.getCols());
        int k=M1.getCols();
        T* d_M1,d_M2,d_M;
        hipMalloc(&d_M1,M1.bytes());
        hipMalloc(&d_M2,M2.bytes());
        hipMalloc(&d_M,M.bytes());
        
    }
};